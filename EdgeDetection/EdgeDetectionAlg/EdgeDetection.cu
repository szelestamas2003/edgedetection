#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <hip/hip_runtime.h>
#include <math.h>
#include "cudaHeader.h"
#include ""

__global__ void gaussianKernel(float* kernel, float sigma, int size) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	float k = (size - 1) / 2;

	int idx = i * blockDim.x + j;
	kernel[idx] = 1.0f / (2 * M_PI * pow(sigma, 2)) * exp(-1 * (pow(i - (k + 1), 2) + pow(j + (k + 1), 2)) / 2 * pow(sigma, 2));
}

__global__ void edgedetection(unsigned char* src, unsigned char* dst, int width, int height, float* kernel) {
	int currY = blockIdx.y * blockDim.y + threadIdx.y;
	int currX = blockIdx.x * blockDim.x + threadIdx.x;


}


unsigned char* launchKernels(unsigned char* pixels, int width, int height) {

	unsigned char* resultPixels = new unsigned char[width * height];
	int kernelSize = 5;
	float sigma = 1.0;

	float* GaussianKernel;
	size_t cudaKernelSize = kernelSize * kernelSize * sizeof(float);

	hipMalloc((void**)&GaussianKernel, cudaKernelSize);

	gaussianKernel << <kernelSize, kernelSize >> > (GaussianKernel, sigma, kernelSize);
	hipDeviceSynchronize();

	unsigned char* cudaSrcPixels;
	unsigned char* cudaDstPixels;

	size_t size = height * width * sizeof(unsigned char);

	hipMalloc((void**)&cudaSrcPixels, size);
	hipMalloc((void**)&cudaDstPixels, size);
	hipMemcpy(cudaSrcPixels, pixels, size, hipMemcpyHostToDevice);

	dim3 block_size(32, 32);
	dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

	edgedetection << <block_size, grid_size >> > (cudaSrcPixels, cudaDstPixels, width, height, GaussianKernel);
	hipDeviceSynchronize();

	hipMemcpy(resultPixels, cudaDstPixels, size, hipMemcpyDeviceToHost);

	hipFree(cudaSrcPixels);
	hipFree(cudaDstPixels);
	hipFree(GaussianKernel);

	return resultPixels;
}